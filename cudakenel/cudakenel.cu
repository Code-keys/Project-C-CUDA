#include "hip/hip_runtime.h"
#include <cudakenel.cuh>

#include "hip/hip_runtime.h"  
#include ""
// #include "hip/device_functions.h"

#include <stdio.h>
#include <malloc.h>
#include <iostream>

#include<boost/timer.hpp>  // test time 

// suanli 7.5
// 定义总数据矩阵的行数和列数  1024 1024  64
// global:主机呼叫，设备执行
// host:主机呼叫，主机执行
// device:设备呼叫，设备执行

// 控制一次计算占用显存的大小：CONfloatROL_ROWS*ROWS*8（字节）< 显存 
// 定义每一块内的线程个数，Gfloat720最多是1024
#define NUM_THREADS 1024

__device__ void gpuhello(void){
    printf("inserted function!\n");
}
__global__ void add( float dev_a[],float dev_b[],float dev_c[],int n)
{
    //dim3 BD = blockDim;
    int x =threadIdx.x;
    int y =threadIdx.y;
    int where = y*blockDim.x+x;
    gpuhello();

    if (where<n){
        for (int i = 0; i<int(blockDim.x/2);++i){
            dev_c[where] = dev_a[i]+dev_b[i];
        }
    }
};

extern "C"
void Interface(float *aa, float *bb, float *cc, int n){
    /*
    aa:输入矩阵 　　内存
    bb:输入矩阵　　内存
    cc:　输出矩阵　内存
    n" 线程数目　或矩阵元素总数　栈
    */
    float* dev_a;
    float* dev_b;
    float* dev_c;
    
    InitCUDA();
    _set_device(0);	
    CUDA_CHECK(hipDeviceReset());

    CUDA_CHECK( hipMalloc((void**)&dev_a, sizeof(float)));
    CUDA_CHECK( hipMalloc((void**)&dev_b, sizeof(float)));
    CUDA_CHECK( hipMalloc((void**)&dev_c, n*sizeof(float)));
    CUDA_CHECK(hipMemcpy(dev_a, aa, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, bb, sizeof(float), hipMemcpyHostToDevice));

    if(dev_c == NULL)exit(0);
    if(dev_b == NULL)exit(0);
    if(dev_a == NULL)exit(0);

    dim3 grid(1, 1);
    dim3 block(32, 32, 1); 

    boost::timer t;
    add<<<grid,block>>>(dev_a,dev_b,dev_c,n);
    std::cout<<"运行时间："<<t.elapsed() <<"　s"<<std::endl;//输出已流失的时间

    CUDA_CHECK(hipMemcpy(cc,dev_c,n*sizeof(float) ,hipMemcpyDeviceToHost));
    CUDA_CHECK (hipFree(dev_a));//释放GPU内存
    CUDA_CHECK (hipFree(dev_b));//释放GPU内存
    CUDA_CHECK (hipFree(dev_c));//释放GPU内存
    CUDA_CHECK (hipDeviceReset());


    for  (int i = 0; i< 1; ++i){
        printf("answer :  %f\n", cc[i]);
    };
    return;
};


int main(int argc, char const *argv[])
{    
    int n = 5;
    int size = sizeof(float);
    float * a =new float;
    float *b = new float;
    float *c = new float[n];
    // float* a = (float*) malloc(  sizeof(float));
    // float* b = (float*) malloc(  sizeof(float));
    // float* c = (float*) malloc(  n* sizeof(float));

    int temp[6] = {0,1,2,3,4,5};

    std:: vector<std::vector<int> > Matrix(5, std::vector<int>(6)); 
    for (std::vector<int> i:Matrix){
        std::copy(temp,temp+6,i.begin());
        std::cout <<i.data()<<std::endl;
    } 

    float *dev_a ;
    float *dev_b ;
    float *dev_c ;

    *a = 1.0;
    *b = 5.0;

    for (int j = 0; j < n; ++j)
    {
        c[j]= 2.0+0.9*j;
    }

    // 准备数据 在栈里   申请主机;内存
    // Choose which GPU to run on, change this on a multi-GPU system.
    // InitCUDA();
    //_set_device(0);	
    // CUDA_CHECK (hipDeviceReset());
    //3.请求CUDA设备的内存（显存），执行CUDA函数   只能访问设备内存
    InitCUDA();
    _set_device(0);	
    CUDA_CHECK (hipDeviceReset());

    CUDA_CHECK ( hipMalloc((void**)&dev_c, n *size));
    CUDA_CHECK ( hipMalloc((void**)&dev_a,size));
    CUDA_CHECK ( hipMalloc((void**)&dev_b, size));
    //4.从主机复制数据到设备上

    if(dev_c == NULL)exit(0);
    if(dev_b == NULL)exit(0);
    if(dev_a == NULL)exit(0);

    CUDA_CHECK(hipMemcpy(dev_a, a, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, b, sizeof(float), hipMemcpyHostToDevice));

    //调用核函数在GPU上执行。数据较少，使用一个Block，含有1024个线程    使用1024个Block  ( 1024*1024*64)   ：：：1400个cuda核心
    //add<<<1,5>>>(dev_a,dev_b,dev_c);
    dim3 grid(1, 1);
    dim3 block(5, 1, 1); 

    add<<<grid,block>>>(dev_a,dev_b,dev_c,n);
   
    // hipDeviceSynchronize();

    CUDA_CHECK(hipMemcpy(c,dev_c,5*sizeof(float) ,hipMemcpyDeviceToHost));
    // CUDA_CHECK (hipDeviceReset());
    CUDA_CHECK (hipFree(dev_a));//释放GPU内存
    CUDA_CHECK (hipFree(dev_b));//释放GPU内存
    CUDA_CHECK (hipFree(dev_c));//释放GPU内存
    //CUDA_CHECK (hipDeviceReset());


    for  (int i = 0; i< 5; ++i){
        printf("answer :  %f\n", c[i]);
    }
    bye("GO !");


    return  0;
}

bool InitCUDA(void)
{
     int count;
     hipGetDeviceCount(&count);
     if (count == 0) {
         printf("floathere is no device.\n");
         return 0;
        }
     int i;
     for (i = 0; i < count; i++) {
         hipDeviceProp_t prop;
         if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
             if (prop.major >= 1) {
                 break;
                }
            }
        if (i == count) {
            printf("floathere is no device supporting CUDA 1.x.\n");
            return 0;
            }
        hipSetDevice(i);
        };
        return  1;
}

void _set_device(int device_id) {
    int current_device;
    CUDA_CHECK(hipGetDevice(&current_device));
    if (current_device == device_id) {
        return;
        }
    // floathe call to hipSetDevice must come before any calls to Get, which
    // may perform initialization using the GPU.
    CUDA_CHECK(hipSetDevice(device_id));
}
